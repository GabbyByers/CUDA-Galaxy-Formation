#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void simulationKernel(Particle* device_particles, int num_particles) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= num_particles)
        return;

    // do physics simulation
}

__global__ void renderKernel(int* device_pixel_particle_counts, unsigned char* device_image_pixels, int num_pixels) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= num_pixels)
        return;

    // determine pixel color
}

void deviceSimulation(Particle* device_particles, int num_particles) {
    unsigned int NUM_THREADS = 512;
    unsigned int NUM_BLOCKS = (num_particles + NUM_THREADS - 1) / NUM_THREADS;
    simulationKernel <<<NUM_BLOCKS, NUM_THREADS>>> (device_particles, num_particles);
    hipDeviceSynchronize();
}

void deviceRenderer(int* device_pixel_particle_counts, unsigned char* device_image_pixels, int num_pixels) {
    unsigned int NUM_THREADS = 512;
    unsigned int NUM_BLOCKS = (num_pixels + NUM_THREADS - 1) / NUM_THREADS;
    renderKernel <<<NUM_BLOCKS, NUM_THREADS>>> (device_pixel_particle_counts, device_image_pixels, num_pixels);
    hipDeviceSynchronize();
}

Particle* allocateDeviceParticles(int num_particles) {
    Particle* device_particles = nullptr;
    hipMalloc((void**)&device_particles, num_particles * sizeof(Particle));
    return device_particles;
}

int* allocateDeviceParticlePositions(int num_particles) {
    int* device_particle_positions = nullptr;
    hipMalloc((void**)&device_particle_positions, num_particles * sizeof(int));
    return device_particle_positions;
}

int* allocateDevicePixelParticleCounts(int num_pixels) {
    int* device_pixel_particle_counts = nullptr;
    hipMalloc((void**)&device_pixel_particle_counts, num_pixels * sizeof(int));
    return device_pixel_particle_counts;
}

unsigned char* allocateDeviceImagePixels(int num_pixels) {
    unsigned char* device_image_pixels = nullptr;
    hipMalloc((void**)&device_image_pixels, num_pixels * 4);
    return device_image_pixels;
}

void freeDeviceParticles(Particle* device_particles) {
    hipFree(device_particles);
}

void freeDeviceParticlePositions(int* device_particle_positions) {
    hipFree(device_particle_positions);
}

void freeDevicePixelParticleCounts(int* device_pixel_particle_counts) {
    hipFree(device_pixel_particle_counts);
}

void freeDeviceImagePixels(unsigned char* device_image_pixels) {
    hipFree(device_image_pixels);
}